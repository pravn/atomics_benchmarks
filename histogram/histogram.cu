#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <Timer.h>
#include <omp.h>

//histogram with N bins in several blocks 
//compute histogram using shared memory atomics
//do a reduction (atomicAdd) in shared memory
//then do a coalesced write to global memory (atomicAdd)

const long int NUM_BLOCKS=256;
#define NUM_BINS 32
#define NUM_THREADS_PER_BLOCK 256


__global__ void shmem_atomics_reducer(int *data, int *count){
  uint tid = blockIdx.x*blockDim.x + threadIdx.x;

  __shared__ int block_reduced[NUM_THREADS_PER_BLOCK];
  block_reduced[threadIdx.x] = 0;

  __syncthreads();

    atomicAdd(&block_reduced[data[tid]],1);
  __syncthreads();

  for(int i=threadIdx.x; i<NUM_BINS; i+=NUM_BINS)
    atomicAdd(&count[i],block_reduced[i]);
  
}
	  

void run_atomics_reducer(int *h_data){
  int *d_data;
  int *h_result_atomics;
  int *d_result_atomics;
  int *h_result;

  hipMalloc((void **) &d_data, NUM_THREADS_PER_BLOCK*NUM_BLOCKS*sizeof(int));
  hipMemcpy(d_data, h_data, NUM_THREADS_PER_BLOCK*NUM_BLOCKS*sizeof(int), hipMemcpyHostToDevice);

  h_result = new int[NUM_BINS];
  memset(h_result, 0, NUM_BINS*sizeof(int));

  hipMalloc((void **) &d_result_atomics, NUM_BINS*sizeof(int));
  hipMemset(d_result_atomics, 0, NUM_BINS*sizeof(int));

  CUDATimer atomics_timer;
  double gpu_time = 0;
  int niter = 10;


  for(int i=0; i<niter; i++){
	  hipMemset(d_result_atomics, 0, NUM_BINS*sizeof(int));
	  atomics_timer.startTimer();
	  shmem_atomics_reducer<<< NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>> (d_data, d_result_atomics);
	  atomics_timer.stopTimer();
	  gpu_time  += atomics_timer.getElapsedTime();
  }


CPUTimer cpu_timer;
double cpu_time = 0;

 int nproc = omp_get_num_procs();

 omp_set_num_threads(nproc);

 std::cout << "Number of CPU cores for use in OpenMP " << nproc << std::endl;

 for(int iter=0; iter<niter; iter++){
   memset(h_result, 0, NUM_BINS*sizeof(int));
   cpu_timer.startTimer();
#pragma omp parallel for   
   for(int i=0; i<NUM_THREADS_PER_BLOCK*NUM_BLOCKS; i++){
     for(int j=0; j<NUM_BINS; j++){
       if(h_data[i]==j){
	 #pragma omp atomic
	 h_result[j]++;
       }
     }
   }
   cpu_timer.stopTimer();
   cpu_time  += cpu_timer.getElapsedTime();
 }


  h_result_atomics = new int[NUM_BINS];
  hipMemcpy(h_result_atomics, d_result_atomics, NUM_BINS*sizeof(int), hipMemcpyDeviceToHost);

  std::cout << "=======================================" << std::endl;
  std::cout << "Computed average atomics bandwidth for " << niter << " iterations " << std::endl;
  std::cout << "Average GPU atomics time in milliseconds " << gpu_time/niter << std::endl;
  std::cout << "Average CPU atomics time in milliseconds " << cpu_time/niter << std::endl;

  float mbytes = NUM_THREADS_PER_BLOCK*NUM_BLOCKS*sizeof(int)*1e-6;

  std::cout << "Megabytes of data " << mbytes << std::endl;

   float gpu_bandwidth = mbytes/gpu_time*niter*1e3;
   float cpu_bandwidth = mbytes/cpu_time*niter*1e3;

   #ifdef WORST_CASE
   std::cout << "Running worst case scenario where all data falls into a single bin " << std::endl;
   #else 
   std::cout << "Running for case where data is distributed randomly into " << NUM_BINS << " bins " << std::endl;
   #endif 

  std::cout << "GPU Atomics bandwidth in MB/s " << gpu_bandwidth << std::endl;
  std::cout << "CPU Atomics bandwidth in MB/s " << cpu_bandwidth << std::endl;
  


  std::cout << "Validation: " << std::endl;
  std::cout << std::setw(4) << "BIN#" 
	    << std::setw(7) << "HOST"
	    << std::setw(9) << "DEVICE" << std::endl;

  for(int i=0; i<NUM_BINS; i++){
    std::cout <<  std::setw(4) << i << " " <<  std::setw(6) << h_result[i] 
	      << " " << std::setw(7) << h_result_atomics[i] << std::endl;
    }

  
  hipFree(d_data);
  delete[] h_result_atomics;
  hipFree(d_result_atomics);
  delete[] h_result;

}
  
  


int main()
{
  int *h_data; 
  h_data = new int[NUM_THREADS_PER_BLOCK*NUM_BLOCKS];

  std::cout << "Data Size " << NUM_THREADS_PER_BLOCK * NUM_BLOCKS << std::endl;

  for(int i=0; i<NUM_THREADS_PER_BLOCK*NUM_BLOCKS; i++){
#ifdef WORST_CASE //worst case scenario when all pixels fall into a single bin
    	   h_data[i] = 0; 
#else
            h_data[i] = (NUM_BINS) * ((float) rand()/RAND_MAX);
#endif
  }

  run_atomics_reducer(h_data);



  //cleanup
  delete[] h_data;

}
